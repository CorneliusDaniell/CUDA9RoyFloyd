#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define N 4

__global__ void RoyFloyd(int a[N][N], int k) {
	int i = threadIdx.x;
	int j = threadIdx.y;

	if (a[i][j] > a[i][k] + a[k][j])
	{
		a[i][j] = a[i][k] + a[k][j];
	}

}

int main(int argc, char **argv) {

	int numBlocks = 1;
	dim3 threadsPerBlock(N, N);
	int size = N*N;

	int a[N][N] = {
		{ 0, 5, 3, 10 },
		{ 6, 0, 3, 4 },
		{ 3, 3, 0, 1 },
		{ 10, 4, 1, 0 }
	};


	int *a_cuda;
	hipMalloc(&a_cuda, size);

	hipMemcpy(a_cuda, a, size, hipMemcpyHostToDevice);

	for (int k = 0; k < N; k++)
	{
		RoyFloyd <<< numBlocks, threadsPerBlock >>> (a_cuda, k);
	}


	hipMemcpy(a, a_cuda, size, hipMemcpyDeviceToHost);
	hipFree(a_cuda);
	return 0;
}